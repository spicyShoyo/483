#include "hip/hip_runtime.h"
#include "stdlib.h"
#include "stdio.h"
#include "math.h"
#define HISTOGRAM_LENGTH 256
#define YSIZE 600
#define XSIZE 1200
#define TrainDataNum 1934
#define DataLen 1024
#define DataWidth 32
#define KernelWidth 5
#define KernelRadius 2
#define TileWidth 8
#define BlockWidth TileWidth+KernelWidth-1
#define RoutingX 79
#define RoutingWidth 512
#define RoutingY 500
#define RoutingHeight 55
#define MoneyX 922
#define MoneyWidth 200
#define MoneyY 218
#define MoneyHeight 64
#define NumX 1010
#define NumWidth 128
#define NumY 53
#define NumHeight 64
#define Boundary 39
#define BLACK_THRESHOLD 80
#define VERIFY_BOUNDARY 15000
#define RED_THRESHOLD 176
#define GREEN_THRESHOLD 176
#define BLUE_THRESHOLD 127
#define Reduced_Data_Length 64
#define KNN_BLOCK_SIZE 32
#define TrainDigits "trainData/foo.csv"
#define TrainLabels "trainData/bar.csv"
#define PCA true

void cuCheck(int line) {
    hipError_t err = hipGetLastError();
    if(err != hipSuccess) {
        printf("Error: %s, %d\n", hipGetErrorString(err), line);
    }
}
//nvcc -arch sm_20 checkReader2.cu

__constant__ float testDigit[DataLen*15];
__constant__ float testPCADigit[Reduced_Data_Length*15];
//image IO code below{{{{{{{{{{{{{{{{{{{{{{{{


void ppmReader(char* fileName, int** container, int* canvasHeight, int* canvasWidth) {
	int width=0;
	int height=0;
	FILE *ptr=fopen(fileName, "r");
	char* buffer=(char*)malloc(sizeof(char)*20);
	fscanf(ptr, "%s", buffer);
	fscanf(ptr, "%chandwriting dataset", buffer);
	while(getc(ptr)!=10) {};
	fscanf(ptr, "%s", buffer);
	width=(int)strtof(buffer, NULL);
	fscanf(ptr, "%c", buffer);
	fscanf(ptr, "%s", buffer);
	height=(int)strtof(buffer, NULL);
	fscanf(ptr, "%c", buffer);
	fscanf(ptr, "%c%c%c%c", buffer, buffer, buffer, buffer);
	*container=(int*)malloc(3*width*height*sizeof(int));
	for(int i=0; i<3*width*height; ++i) {
		fscanf(ptr, "%s", buffer);
		(*container)[i]=(int)(strtof(buffer, NULL));
		fscanf(ptr, "%c", buffer);
	}
	fclose(ptr);
	*canvasWidth=width;
	*canvasHeight=height;
	free(buffer);
}


//output black/white in gray scale
void monoWritterP2(int* image, int ySize, int xSize, char* fileName) {
	FILE *ptr=fopen(fileName, "w");
	fprintf(ptr, "P2\n");
	fprintf(ptr, "%d %d\n", xSize, ySize);
	fprintf(ptr, "255\n");
	for(int i=0; i<ySize*xSize; ++i) {
		fprintf(ptr, "%d\n", 255*(1-image[i]));
	}
	return;
}


void ppmWritter(char* fileName, int* container, int canvasHeight, int canvasWidth) {
	FILE *ptr=fopen(fileName, "w");
	fprintf(ptr, "P3\n");
	fprintf(ptr, "%d %d\n", canvasWidth, canvasHeight);
	fprintf(ptr, "255\n");
	for(int i=0; i<canvasHeight*canvasWidth*3; ++i) {
		fprintf(ptr, "%d\n", container[i]);
	}
	return;
}

//output black/white in black/white
void monoWritter(int* image, int ySize, int xSize, char* fileName) {
	FILE *ptr=fopen(fileName, "w");
	fprintf(ptr, "P1\n");
	fprintf(ptr, "%d %d\n", xSize, ySize);
	for(int i=0; i<ySize*xSize; ++i) {
		fprintf(ptr, "%d\n", image[i]);
	}
	return;
}


void outputImage(int* imageDevice, int ySize, int xSize, char* fileName="test.pgm") {
	int imageSize=sizeof(int)*ySize*xSize;
	int* imageHost=(int*)malloc(imageSize);
	hipMemcpy(imageHost, imageDevice, imageSize, hipMemcpyDeviceToHost);
	cuCheck(__LINE__);
	monoWritter(imageHost, ySize, xSize, fileName);
	free(imageHost);
	return;
}


void printFirstDigit(float* datas, int h=32, int w=32) {
	for(int i=0; i<h; i++) {
		for(int j=0; j<w; j++) {
			printf("%d", (int)datas[i*w+j]);
		}
		printf("\n");
	}
	printf("---------------\n");
	return;
}
//image IO code above}}}}}}}}}}}}}}}}}}}}}}}


//knn code below{{{{{{{{{{{{{{{{{{{{{{{{
int* digits=NULL;
float* trainDataHost=NULL;
float* trainDataKernel=NULL;
float* distantHost=NULL;
float* distantKernel=NULL;


__global__ void knn(float* trainDataKernel, float* distantKernel, int count) {
	__shared__ float digit[DataWidth][DataWidth];
	__shared__ float train[DataWidth][DataWidth];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;

	train[tx][ty]=trainDataKernel[bx*DataLen+ty*DataWidth+tx];

	for(int i=0; i<count; ++i) {
		digit[tx][ty]=testDigit[i*DataLen+ty*DataWidth+tx];
		__syncthreads();

		float cur=digit[tx][ty]-train[tx][ty];
		cur=cur*cur;
		digit[tx][ty]=cur;
		for(int stride=16; stride>0; stride/=2) {
			__syncthreads();
			if(tx<stride&&ty<stride) {
				digit[tx][ty]+=digit[tx+stride][ty]+digit[tx+stride][ty+stride]+digit[tx][ty+stride];
			}
		}
		__syncthreads();
		if(tx==0) {
			distantKernel[TrainDataNum*i+bx]=int(digit[0][0])+float(bx)/10000;
		}

		__syncthreads();
	}
}


// input: the pointer to hold the training datas
// return: return by pointer of the training datas
void initTrainDataHost(float* container) {
	FILE* ptr=fopen(TrainDigits, "r");
	char* buffer=(char*)malloc(sizeof(char));
	for(int j=0; j<TrainDataNum; ++j) { 
		for(int i=0; i<DataLen; ++i) {
			fscanf(ptr, "%c", buffer);
			container[j*DataLen+i]=(float)(buffer[0])-48;
		}
		fscanf(ptr, "%c", buffer);
	}
	free(buffer);
	return;
}

void initEigenvectors(float* container) {
	FILE* ptr=fopen("eigenvectors.csv", "r");
	char* buffer=(char*)malloc(sizeof(char));
	for(int j=0; j<DataLen; ++j) {
		for(int i=0; i<Reduced_Data_Length; ++i) {
			fscanf(ptr, "%f", &container[j*Reduced_Data_Length+i]);
			fscanf(ptr, "%c", buffer);
		}
	}
	free(buffer);
	return;
}

//init the answer to the training data
//return by pointer
//this is because the training data only holds the data
//it self, so here is what the data is.
void initDigits(int* container) {
	FILE* ptr=fopen(TrainLabels, "r");
	char* buffer=(char*)malloc(sizeof(char));
	for(int i=0; i<TrainDataNum; ++i) {
		fscanf(ptr, "%c", buffer);
		container[i]=(int)(buffer[0])-48;
		fscanf(ptr, "%c", buffer);
	}
	free(buffer);
	return;
}


void initKNN() {
	digits=(int*)malloc(TrainDataNum*sizeof(int));
	initDigits(digits);

	int trainDataSize=sizeof(float)*DataLen*TrainDataNum;
	trainDataHost=(float*)malloc(trainDataSize);
	initTrainDataHost(trainDataHost);
	hipMalloc((void**) &trainDataKernel, trainDataSize);
	cuCheck(__LINE__);
	hipMemcpy(trainDataKernel, trainDataHost, trainDataSize, hipMemcpyHostToDevice);
	cuCheck(__LINE__);

	int distantSize=sizeof(float)*TrainDataNum*15;
	distantHost=(float*)malloc(distantSize);
	hipMalloc((void**) &distantKernel, distantSize);
	cuCheck(__LINE__);
}


void freeKNN() {
	free(digits);
	hipFree(trainDataKernel);
	free(trainDataHost);
	free(distantHost);
	hipFree(distantKernel);
}


//merge sort helper
void merge(float* distantHost, int n, int m) {
	int i, j, k;
	float* x=(float*)malloc(n*sizeof(float));
	// int* y=(int*)malloc(n*sizeof(int));
	for(i=0, j=m, k=0; k<n; k++) {
		if(j==n) {
			x[k]=distantHost[i];
			// y[k]=digits[i];
			i+=1;
		}else if(i==m) {
			x[k]=distantHost[j];
			// y[k]=digits[j];
			j+=1;
		}else if(int(distantHost[j])<int(distantHost[i])) {
			x[k]=distantHost[j];
			// y[k]=digits[j];
			j+=1;
		}else {
			x[k]=distantHost[i];
			// y[k]=digits[i];
			i+=1;
		}
	}
	for(int i=0; i<n; i++) {
		distantHost[i]=x[i];
		// digits[i]=y[i];
	}
	free(x);
	// free(y);
}


//sort the output array from knn
void mergeSort(float* distantHost, int n) {
	if(n<2) {
		return;
	}
	int m=n/2;
	mergeSort(distantHost, m);
	mergeSort(distantHost+m, n-m);
	merge(distantHost, n, m);
}


void recognizePCA(int* ans, int count) {
	int distantSize=sizeof(float)*TrainDataNum*count;
	dim3 dimBlock(Reduced_Data_Length, 1, 1);
	dim3 dimGrid(TrainDataNum, 1, 1);

	knnPCA<<<dimGrid, dimBlock>>>
}


void recognize(int* ans, int count) {
	int distantSize=sizeof(float)*TrainDataNum*15;
	dim3 dimBlock(DataWidth, DataWidth, 1);
	dim3 dimGrid(TrainDataNum, 1, 1);
	
	knn<<<dimGrid, dimBlock>>>(trainDataKernel, distantKernel, count);
	
	hipDeviceSynchronize();
	cuCheck(__LINE__);
	hipMemcpy(distantHost, distantKernel, distantSize, hipMemcpyDeviceToHost);
	cuCheck(__LINE__);
	for(int j=0; j<count; ++j) {
		float* curDistantHost=distantHost+j*TrainDataNum;
		mergeSort(curDistantHost, TrainDataNum);
		int num[10]={};
		if(curDistantHost[0]>180) {
			ans[j]=-1;
			continue;
		}
		for(int i=0; i<12; i++) {
			num[digits[int(10000*(curDistantHost[i]-int(curDistantHost[i])))]]+=1;
		}
		int curBest=-1;
		int curInt=-1;
		for(int i=0; i<10; i++) {
			if(num[i]!=0&&num[i]>curBest) {
				curBest=num[i];
				curInt=i;
			}
		}
		ans[j]=curInt;
	}
}


//noise reduction code below{{{{{{{{{{{{{{{{{{{{{{{{
__constant__ float Mask[KernelWidth][KernelWidth];

//this one is convolution in 2d for reduce noise
__global__ void conv2d(int* A, int* B, const int ySize, const int xSize) {
	__shared__ float sharedM[BlockWidth][BlockWidth];
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int colO=blockIdx.x*TileWidth+threadIdx.x;
	int rowO=blockIdx.y*TileWidth+threadIdx.y;
	int colI=colO-KernelRadius;
	int rowI=rowO-KernelRadius;

	if((colI>=0)&&(colI<xSize)&&(rowI>=0)&&(rowI<ySize)) {
		sharedM[tx][ty]=A[rowI*xSize+colI];
	}else {
		sharedM[tx][ty]=0;
	}
	__syncthreads();

	float output=0;
	if(tx<TileWidth&&ty<TileWidth) {
		for(int y=0; y<KernelWidth; ++y) {
			for(int x=0; x<KernelWidth; ++x) {
				output+=Mask[x][y]*sharedM[tx+x][ty+y];
			}
		}
	}
	output=output>15? 1:0;
	if(rowO<ySize&&colO<xSize&&tx<TileWidth&&ty<TileWidth) {
		B[rowO*xSize+colO]=output;
	}
}


//reduce the noise of the canvas
void noiseReduct(int* digit, int ySize, int xSize) {
	int maskSize=sizeof(float)*KernelWidth*KernelWidth;
	float* maskHost=(float*)malloc(maskSize);
	for(int i=0; i<KernelWidth*KernelWidth; ++i) {
		maskHost[i]=1;
	}
	hipMemcpyToSymbol(HIP_SYMBOL(Mask), maskHost, maskSize);
	free(maskHost);

	int digitSize=ySize*xSize*sizeof(int);
	
	int* digitInput;
	int* digitOutput;
	digitInput=digit;
	hipMalloc((void**) &digitOutput, digitSize);
	cuCheck(__LINE__);
	dim3 dimBlock(BlockWidth, BlockWidth, 1);
	dim3 dimGrid(ceil(xSize/(float)TileWidth), ceil(ySize/(float)TileWidth), 1);
	conv2d<<<dimGrid, dimBlock>>>(digitInput, digitOutput, ySize, xSize);
	hipDeviceSynchronize();
	cuCheck(__LINE__);
	hipMemcpy(digit, digitOutput, digitSize, hipMemcpyDeviceToHost);
	cuCheck(__LINE__);
	hipFree(digitOutput);
}
//noise reduction code above}}}}}}}}}}}}}}}}}}}}}}}


//strip elimination below{{{{{{{{{{{{{{{{{{{{{{{{
__global__ void stripEliminationDevice(int* checkMonoDevice, int* outDevice, int ySize=YSIZE, int xSize=XSIZE) {
	__shared__ int partialSum[1024];
	unsigned int tx=threadIdx.x;
	unsigned int ty=blockIdx.y;
	unsigned int start=2*blockIdx.x*blockDim.x;
	partialSum[tx]=0;
	if(start+tx<xSize) {
		partialSum[tx]=checkMonoDevice[ty*(xSize)+start+tx];
	}
	if(start+blockDim.x+tx<xSize) {
		partialSum[tx]+=checkMonoDevice[ty*(xSize)+start+blockDim.x+tx];
	}
	for(unsigned int stride=blockDim.x/2; stride>0; stride/=2) {
		__syncthreads();
		if(tx<stride) {
			partialSum[tx]+=partialSum[tx+stride];
		}
	}
	__syncthreads();
	if(tx==0) {
		outDevice[ty]=partialSum[tx];
	}
	if(partialSum[0]>500) {
		if(ty*(xSize)+start+tx<xSize*ySize) {
			checkMonoDevice[ty*(xSize)+start+tx]=0;
		}
		if(ty*(xSize)+start+blockDim.x+tx<xSize*ySize) {
			checkMonoDevice[ty*(xSize)+start+blockDim.x+tx]=0;
		}
	}
}


void stripEliminationHost(int* checkMonoDevice, int ySize, int xSize) {
	int* outHost;
	int* outDevice;
	int outSize=sizeof(int)*ySize;
	outHost=(int*)malloc(outSize);
	hipMalloc((void **) &outDevice, outSize);

	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(1, ySize, 1);
	stripEliminationDevice<<<dimGrid, dimBlock>>>(checkMonoDevice, outDevice, ySize, xSize);
	hipDeviceSynchronize();
	cuCheck(__LINE__);

	hipMemcpy(outHost, outDevice, outSize, hipMemcpyDeviceToHost);
	free(outHost);
	hipFree(outDevice);

	return;
}
//strip elimination code above}}}}}}}}}}}}}}}}}}}}}}}


//scale code below{{{{{{{{{{{{{{{{{{{{{{{{
void scaleHost(int** checkColoredDevice, int* image, int ySize, int xSize) {
	//alloc memory
	int checkColoredSize=sizeof(int)*3*ySize*xSize;
	hipMalloc((void **) checkColoredDevice, checkColoredSize);
	cuCheck(__LINE__);

	//scale, assume no need to scale now
	hipMemcpy(*checkColoredDevice, image, checkColoredSize, hipMemcpyHostToDevice);
	cuCheck(__LINE__);

	return;
}
//scale code above}}}}}}}}}}}}}}}}}}}}}}}


//verification cuda below{{{{{{{{{{{{{{{{{{{{{{{{

__global__ void verifyBlue(int* input_arr, int* isBlue) {
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int input_index=row*XSIZE+col;
	int output_index=row*2048+col;
	if (row<YSIZE && col<XSIZE && input_arr[3*input_index]<RED_THRESHOLD && input_arr[3*input_index+1]<GREEN_THRESHOLD && input_arr[3*input_index+2]>BLUE_THRESHOLD)
		isBlue[output_index]=1;
	else
		isBlue[output_index]=0;
	return;
}

__global__ void countBlue_perRow(int* isBlue, int* numBlue_perRow) {
	__shared__ int partial[2048];
	int BLOCK_SIZE=1024;
	int by=blockIdx.y;
	int tx=threadIdx.x;
	partial[tx]=isBlue[by*2*BLOCK_SIZE+tx];
	partial[BLOCK_SIZE+tx]=isBlue[by*2*BLOCK_SIZE+BLOCK_SIZE+tx];
	__syncthreads();
	for (int stride=BLOCK_SIZE; stride>0; stride/=2)
	{
		if (tx<stride)
			partial[tx]+=partial[tx+stride];
		__syncthreads();
	}
	if (tx==by)
		numBlue_perRow[tx]=partial[0];
	return;
}

__global__ void countBlue_total(int* numBlue_perRow, int* total) {
	__shared__ int partial[1024];
	int BLOCK_SIZE=512;
	int bx=blockIdx.x;
	int tx=threadIdx.x;
	partial[tx]=numBlue_perRow[bx*2*BLOCK_SIZE+tx];
	partial[BLOCK_SIZE+tx]=numBlue_perRow[bx*2*BLOCK_SIZE+BLOCK_SIZE+tx];
	__syncthreads();
	for (int stride=BLOCK_SIZE; stride>0; stride/=2)
	{
		if (tx<stride)
			partial[tx]+=partial[tx+stride];
		__syncthreads();
	}
	if (tx==bx)
		total[tx]=partial[0];
	return;
}

int verificationHost(int* checkColoredDevice, int ySize=YSIZE, int xSize=XSIZE) {
	int* device_check;
	int* isBlue;
	int* numBlue_perRow;
	int* host_total;
	int* device_total;
	int size=ySize*xSize;

	hipMalloc((void**)&device_check, 3*size*sizeof(int));
	hipMalloc((void**)&isBlue, 2048*1024*sizeof(int));

	hipMemcpy(device_check, checkColoredDevice, 3*size*sizeof(int), hipMemcpyHostToDevice);
	cuCheck(__LINE__);

	dim3 DimGrid1(2, 1024, 1);
	dim3 DimBlock1(1024, 1, 1);
	verifyBlue<<<DimGrid1, DimBlock1>>>(device_check, isBlue);
	hipFree(device_check);
	cuCheck(__LINE__);

	hipMalloc((void**)&numBlue_perRow, 1024*sizeof(int));
	host_total=(int*)malloc(sizeof(int));
	hipMalloc((void**)&device_total, sizeof(int));
	cuCheck(__LINE__);

	dim3 DimGrid2(1, 1024, 1);
	dim3 DimBlock2(1024, 1, 1);
	countBlue_perRow<<<DimGrid2, DimBlock2>>>(isBlue, numBlue_perRow);
	hipFree(isBlue);
	cuCheck(__LINE__);

	dim3 DimGrid3(1, 1, 1);
	dim3 DimBlock3(512, 1, 1);
	countBlue_total<<<DimGrid3, DimBlock3>>>(numBlue_perRow, device_total);
	hipFree(numBlue_perRow);
	cuCheck(__LINE__);

	hipMemcpy(host_total, device_total, sizeof(int), hipMemcpyDeviceToHost);
	cuCheck(__LINE__);

	int total=*host_total;

	free(host_total);
	hipFree(device_total);

	printf("number of blue pixels are %d\n", total);

	if (total>size/8 && total<size/2)
		return 1;
	return 0;
}
//verification code above}}}}}}}}}}}}}}}}}}}}}}}


//convert to mono code below{{{{{{{{{{{{{{{{{{{{{{{{
__global__ void toMonoDevice(int* checkMonoDevice, int* checkColoredDevice, int ySize, int xSize) {
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<ySize*xSize) {
		int r=checkColoredDevice[3*index];
		int g=checkColoredDevice[3*index+1];
		int b=checkColoredDevice[3*index+2];
		checkMonoDevice[index]=((r+g+b)/3)>100? 0:1;
	}
}


void toMonoHost(int** checkMonoDevice, int* checkColoredDevice, int ySize=YSIZE, int xSize=XSIZE) {
	//alloc memory
	int checkMonoSize=sizeof(int)*ySize*xSize;
	hipMalloc((void **) checkMonoDevice, checkMonoSize);
	cuCheck(__LINE__);

	//convert to gray here
	dim3 dimBlock(1024, 1, 1);
	dim3 dimGrid(ceil(xSize*ySize/(float)1024), 1, 1);
	toMonoDevice<<<dimGrid, dimBlock>>>(*checkMonoDevice, checkColoredDevice, ySize, xSize);
	hipDeviceSynchronize();
	cuCheck(__LINE__);

	// outputImage(*checkMonoDevice, ySize, xSize, "checkMono.pgm");

	return;
}
//convert to mono code above}}}}}}}}}}}}}}}}}}}}}}}


//grab and read digit code below{{{{{{{{{{{{{{{{{{{{{{{{
__global__ void getHorizonDevice(int* horizonDevice, int* checkMonoDevice, int grabX, int grabY, int grabWidth, int grabHeight, int ySize=YSIZE, int xSize=XSIZE) {
	int bx=blockIdx.x;
	int inX=bx+grabX;
	int inY=grabY;
	int ans=0;
	for(int i=0; i<grabHeight; ++i) {
		ans+=checkMonoDevice[(i+inY)*xSize+inX];
	}
	horizonDevice[bx]=ans;
}


void getAns(int* ans, int count) {
	recognize(ans, count);
	return;
}


__global__ void matrixMultiply_device(float *A, float *B, float *C, int numARows, int numAColumns, int numBRows, int numBColumns, int numCRows, int numCColumns) {
	__shared__ float sharedA[32][32];
	__shared__ float sharedB[32][32];
	int Row=blockIdx.y*32+threadIdx.y;
	int Column=blockIdx.x*32+threadIdx.x;
	float resultC=0.0f;
	for (int i=0; i<ceil(numAColumns/32.0f); i++)
	{
		if (Row<numARows && i*32+threadIdx.x<numAColumns)
			sharedA[threadIdx.y][threadIdx.x]=A[Row*numAColumns+i*32+threadIdx.x];
		else
			sharedA[threadIdx.y][threadIdx.x]=0.0f;
		if (i*32+threadIdx.y<numBRows && Column<numBColumns)
			sharedB[threadIdx.y][threadIdx.x]=B[(i*32+threadIdx.y)*numBColumns+Column];
		else
			sharedB[threadIdx.y][threadIdx.x]=0.0f;
		__syncthreads();
		for (int j=0; j<32; j++)
		{
			resultC+=sharedA[threadIdx.y][j]*sharedB[j][threadIdx.x];
		}
		__syncthreads();
	}
	if (Row<numCRows && Column<numCColumns)
		C[Row*numCColumns+Column]=resultC;
	return;
}


void setPCAConstant(float* digitHost, int count) {
	int digitSize=sizeof(float)*count*DataLen);
	int digitPCADSize=sizeof(float)*count*Reduced_Data_Length;
	float* digitDevice;
	float* digitPCADevice;
	hipMalloc((void **) &digitDevice, digitSize);
	hipMalloc((void **) &digitPCADevice, digitPCADSize);
	hipMemcpy(digitDevice, digitHost, digitSize, hipMemcpyHostToDevice);
	cuCheck(__LINE__);

	dim3 DimGrid(ceil(Reduced_Data_Length/32.0f),ceil(count/32.0f),1);
	dim3 DimBlock(32,32,1);
	matrixMultiply_device<<<dimGrid, dimBlock>>>(digitDevice, EigenVectors, digitPCADevice, count, DataLen, DataLen, Reduced_Data_Length, count, Reduced_Data_Length);
	hipDeviceSynchronize();
	cuCheck(__LINE__);

	hipMemcpyToSymbol(HIP_SYMBOL(testPCADigit), digitPCADevice, Reduced_Data_Length*count);
	cuCheck(__LINE__);
	hipFree(digitDevice);
	hipFree(digitPCADevice);
}


int* readAreaHost(int* checkMonoDevice, int grabX, int grabY, int grabWidth, int grabHeight, int num, int ySize=YSIZE, int xSize=XSIZE) {
	//outputImage(checkMonoDevice, ySize, xSize, "area.pgm");
	int count=0;
	int checkMonoSize=sizeof(int)*ySize*xSize;
	int* checkMonoHost=(int*)malloc(checkMonoSize);
	hipMemcpy(checkMonoHost, checkMonoDevice, checkMonoSize, hipMemcpyDeviceToHost);
	cuCheck(__LINE__);
	//15 is the length of routing number + account number
	int* ans=(int*)malloc(sizeof(int)*15);
	//set all to -1 as init state
	for(int i=0; i<15; ++i) {
		ans[i]=-1;
	}

	int horizonSize=sizeof(int)*grabWidth;
	int* horizonHost;
	int* horizonDevice;
	horizonHost=(int*)malloc(horizonSize);
	hipMalloc((void **) &horizonDevice, horizonSize);

	dim3 dimBlock(1, 1, 1);
	dim3 dimGrid(grabWidth, 1, 1);
	getHorizonDevice<<<dimGrid, dimBlock>>>(horizonDevice, checkMonoDevice, grabX, grabY, grabWidth, grabHeight, ySize, xSize);
	hipDeviceSynchronize();
	cuCheck(__LINE__);
	//outputImage(horizonDevice, 1, grabWidth);
	hipMemcpy(horizonHost, horizonDevice, horizonSize, hipMemcpyDeviceToHost);
	hipFree(horizonDevice);
	cuCheck(__LINE__);
	float* digitHost=(float*)malloc(sizeof(float)*DataLen*15);
	int testDigitSize=sizeof(float)*DataLen;
	int left=0;
	int right=0;
	while(left<grabWidth&&count<num) {
		if(horizonHost[left]==0) {
			++left;
		}else {
			right=left;
			while(right<grabWidth&&horizonHost[right]!=0) {
				++right;
			}

			int midX=(left+right)/2+grabX;
			int top=-1;
			int bottom=-1;
			//printf("%d, %d\n", left, right);
			for(int y=0; (y<grabHeight/2&&((top==-1)||(bottom==-1))); ++y) {
				int yt=y+grabY;
				int yb=grabY+grabHeight-y;
				for(int xi=left+grabX; xi<right+grabX; ++xi) {
					if(top==-1&&checkMonoHost[yt*xSize+xi]!=0) {
						top=yt;
					}
					if(bottom==-1&&checkMonoHost[yb*xSize+xi]!=0) {
						bottom=yb;
					}
				}
			}
			int midY=(top+bottom)/2;
			int startX=midX-16;
			int startY=midY-16;
			for(int i=startY; i<startY+DataWidth; ++i) {
				for(int j=startX; j<startX+DataWidth; ++j) {
					int x=j-startX;
					int y=i-startY;
					digitHost[count*DataLen+y*DataWidth+x]=(float)checkMonoHost[i*xSize+j];
				}
			}
			++count;
			left=right;
		}

	}

	if(PCA) {
		setPCAConstant(digitHost, count);
		getPCAAns(ans, count);
		recognizePCA(ans, count);
	}else {
		hipMemcpyToSymbol(HIP_SYMBOL(testDigit), digitHost, testDigitSize*15);
		cuCheck(__LINE__);
		getAns(ans, count);
	}
	free(digitHost);
	free(checkMonoHost);	
	return ans;
}
//grab and read digit code above}}}}}}}}}}}}}}}}}}}}}}}


//read check code below{{{{{{{{{{{{{{{{{{{{{{{{
void checkReaderHost(int* checkMonoDevice, int ySize=YSIZE, int xSize=XSIZE) {
	stripEliminationHost(checkMonoDevice, ySize, xSize);

	//grab the area left, but later I found that don't really need to.
	int* ans;

	ans=readAreaHost(checkMonoDevice, RoutingX, RoutingY, RoutingWidth, RoutingHeight, 15, ySize, xSize);
	
	char routing[9];
	for(int i=0; i<9; ++i) {
		routing[i]=char(ans[i]+48);
	}
	char account[6];
	for(int i=9; i<15; ++i) {
		account[i-9]=char(ans[i]+48);
	}
	free(ans);
	printf("Routing Number: %s\n", routing);
	printf("Account Number: %s\n", account);

	ans=readAreaHost(checkMonoDevice, MoneyX, MoneyY, MoneyWidth, MoneyHeight, 6, ySize, xSize);
	float money=0;
	int j=0;
	while(ans[j]!=-1) {
		money=money*10+ans[j];
		j++;
	}
	money=money/100;
	free(ans);
	printf("Amount: %.2f\n", money);

	char num[3];
	ans=readAreaHost(checkMonoDevice, NumX, NumY, NumWidth, NumHeight, 3, ySize, xSize);
	for(int i=0; i<3; ++i) {
		num[i]=char(ans[i]+48);
	}
	free(ans);
	printf("Check Number: %s\n", num);

	return;
}
//read check code above}}}}}}}}}}}}}}}}}}}}}}}


void readSingleCheck(int* in) {
	//read in the image
	int* imageHost=NULL;
	int* ySize=(int*)malloc(sizeof(int));
	int* xSize=(int*)malloc(sizeof(int));

	int* checkColoredDevice=NULL;
	
	*xSize=1200;
	*ySize=600;
	imageHost=(int*)malloc(sizeof(int));
	checkColoredDevice=in;

	//verify check
	int valid=verificationHost(checkColoredDevice);
	if(!valid){
	 	printf("Invalid check\n");
	 	return;
	}
	else
	 	printf("Valid Check from Chase Bank\n");

	//convert to mono
	int* checkMonoDevice=NULL;
	toMonoHost(&checkMonoDevice, checkColoredDevice);

	//read the check
	checkReaderHost(checkMonoDevice);

	//free memory
	free(imageHost);
	free(ySize);
	free(xSize);
	hipFree(checkColoredDevice);
	hipFree(checkMonoDevice);
	return;
}

bool isBlack_pixel(int x, int y, int *image_container, int width){
	int index=3*(y*width+x);
	int red=image_container[index];
	int green=image_container[index+1];
	int blue=image_container[index+2];
	return (red<BLACK_THRESHOLD && green<BLACK_THRESHOLD && blue<BLACK_THRESHOLD);
}

bool isBlack(int* container, int x, int y, int ySize, int xSize) {
	int sum=0;
	for(int i=-1; i<2; i+=1) {
		for(int j=-1; j<2; j+=1) {
			int xi=x+i;
			int yi=y+j;
			if(xi>=0 && xi<xSize && yi>=0 && yi<ySize)
				if(isBlack_pixel(xi, yi, container, xSize))
					sum+=1;
		}
	}
	if(sum>4) return true;
	return false;
}

bool width_greater_than_height(int *upperleft, int *upperright, int *lowerleft, int *lowerright){
	int curwidth=(upperleft[0]-upperright[0])*(upperleft[0]-upperright[0])+(upperleft[1]-upperright[1])*(upperleft[1]-upperright[1]);
	int curheight=(upperleft[0]-lowerleft[0])*(upperleft[0]-lowerleft[0])+(upperleft[1]-lowerleft[1])*(upperleft[1]-lowerleft[1]);
	return (curwidth>curheight);
}

int getNewX(int x, int y, int dir){
	if (dir==0) return x+1;
	else if (dir==2) return x-1;
	else return x;
}

int getNewY(int x, int y, int dir){
	if (dir==1) return y+1;
	else if (dir==3) return y-1;
	else return y;
}

void enqueue(int * queue, int value, int * head, int * tail, int maxSize)
{
    if ((*tail - maxSize) == *head)
    {
        printf("Queue is full\n");
        return;
    }
    *tail = *tail + 1;
    queue[*tail % maxSize] = value;
}

int dequeue(int * queue, int * head, int * tail, int maxSize)
{
    if (*head == *tail)
    {
        printf("Queue is empty\n");
        return -1;
    }
    *head = *head + 1;
    return queue[*head % maxSize];
}


void bfs(int *image_container, int *upperleft, int *upperright, int *lowerleft, int *lowerright, int *check_width, int *check_height, int width, int height){
	upperleft[0]=width-1;
	upperleft[1]=height-1;
	upperright[0]=0;
	upperright[1]=height-1;
	lowerleft[0]=width-1;
	lowerleft[1]=0;
	lowerright[0]=0;
	lowerright[1]=0;
	int cur_coord_y=0;
	while (!isBlack(image_container, width/2, cur_coord_y, height, width))
		cur_coord_y+=1;
    int head_x=0;
    int tail_x=0;
    int head_y=0;
    int tail_y=0;
    int queueSize=width*height;
    int* coordinates_x=(int*)malloc(queueSize*sizeof(int));
    int* coordinates_y=(int*)malloc(queueSize*sizeof(int));
    int* visited=(int*)malloc(queueSize*sizeof(int));
    int i;
    int j;
    for (i=0; i<height; i++)
    	for (j=0; j<width; j++)
    		visited[i*width+j]=0;
    enqueue(coordinates_x, width/2, &head_x, &tail_x, queueSize);
    enqueue(coordinates_y, cur_coord_y, &head_y, &tail_y, queueSize);
	while (!(head_x==tail_x && head_y==tail_y))
	{
		int X=dequeue(coordinates_x, &head_x, &tail_x, queueSize);
		int Y=dequeue(coordinates_y, &head_y, &tail_y, queueSize);
		if (X<upperleft[0]){
			upperleft[0]=X;
			upperleft[1]=Y;
		}
		if (X>lowerright[0]){
			lowerright[0]=X;
			lowerright[1]=Y;
		}
		if (Y<upperright[1]){
			upperright[0]=X;
			upperright[1]=Y;
		}
		if (Y>lowerleft[1]){
			lowerleft[0]=X;
			lowerleft[1]=Y;
		} 
		for (int dir=0; dir<4; dir++)
		{
			int newX=getNewX(X, Y, dir);
			int newY=getNewY(X, Y, dir);
			if (newX>=0 && newX<width && newY>=0 && newY<height && isBlack(image_container, newX, newY, height, width))
			{
				if (visited[newY*width+newX]==0)
				{
					visited[newY*width+newX]=1;
					enqueue(coordinates_x, newX, &head_x, &tail_x, queueSize);
					enqueue(coordinates_y, newY, &head_y, &tail_y, queueSize);
				}
			}
		}
	}
	free(coordinates_x);
	free(coordinates_y);
	free(visited);

	if (!width_greater_than_height(upperleft, upperright, lowerleft, lowerright)){ //to be tested
		int x=upperleft[0];
		int y=upperleft[1];
		upperleft[0]=upperright[0];
		upperleft[1]=upperright[1];
		upperright[0]=lowerright[0];
		upperright[1]=lowerright[1];
		lowerright[0]=lowerleft[0];
		lowerright[1]=lowerleft[1];
		lowerleft[0]=x;
		lowerleft[1]=y;
	}
	*check_width=sqrt((upperleft[0]-upperright[0])*(upperleft[0]-upperright[0])+(upperleft[1]-upperright[1])*(upperleft[1]-upperright[1]));
	*check_height=sqrt((upperleft[0]-lowerleft[0])*(upperleft[0]-lowerleft[0])+(upperleft[1]-lowerleft[1])*(upperleft[1]-lowerleft[1]));
	*check_width=*check_width/(float)(XSIZE-2*Boundary)*XSIZE;
	*check_height=*check_height/(float)(YSIZE-2*Boundary)*YSIZE;
}

__global__ void rotation(int* device_input, int* device_output, int input_height, int input_width, int output_height, int output_width, int center_x, int center_y, float cos_theta, float sin_theta){
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int dx=blockDim.x;
	int dy=blockDim.y;
	int output_coord_x=bx*dx+tx;
	int output_coord_y=by*dy+ty;
	int normal_output_x=output_coord_x-output_width/2;
	int normal_output_y=output_coord_y-output_height/2;
	int output_index=output_coord_y*output_width+output_coord_x;
	if (output_coord_x<output_width && output_coord_y<output_height){
		int normal_input_x=cos_theta*normal_output_x+sin_theta*normal_output_y;
		int normal_input_y=cos_theta*normal_output_y-sin_theta*normal_output_x;
		int input_coord_x=normal_input_x+center_x;
		int input_coord_y=normal_input_y+center_y;
		int input_index=input_coord_y*input_width+input_coord_x;
		device_output[3*output_index]=device_input[3*input_index];
		device_output[3*output_index+1]=device_input[3*input_index+1];
		device_output[3*output_index+2]=device_input[3*input_index+2];
	}
}


float getCos(int x1, int y1, int x2, int y2){
	int a=x2-x1;
	int b=y1-y2;
	return (float)a/sqrt((float)(a*a+b*b));
}


float getSin(int x1, int y1, int x2, int y2){
	int a=x2-x1;
	int b=y1-y2;
	return (float)b/sqrt((float)(a*a+b*b));
}


__global__ void resize(int* input, int* output, int input_height, int input_width, int output_height, int output_width){
	int tx=threadIdx.x;
	int ty=threadIdx.y;
	int bx=blockIdx.x;
	int by=blockIdx.y;
	int dx=blockDim.x;
	int dy=blockDim.y;
	int output_coord_x=bx*dx+tx;
	int output_coord_y=by*dy+ty;
	int output_index=output_coord_y*output_width+output_coord_x;
	if (output_coord_x<output_width && output_coord_y<output_height){
		int input_coord_x=output_coord_x*input_width/output_width;
		int input_coord_y=output_coord_y*input_height/output_height;
		int input_index=input_coord_y*input_width+input_coord_x;
		output[3*output_index]=input[3*input_index];
		output[3*output_index+1]=input[3*input_index+1];
		output[3*output_index+2]=input[3*input_index+2];
	}
}


int* preprocess(char* fileName){
	int check_width=0;
	int check_height=0;
	int upperleft[2];
	int upperright[2];
	int lowerleft[2];
	int lowerright[2];
	int height=0;
	int width=0;
	int* input_image;
	int* output_resized_image;

	ppmReader(fileName, &input_image, &height, &width);

	bfs(input_image, upperleft, upperright, lowerleft, lowerright, &check_width, &check_height, width, height);
	printf("upperleft= %d, %d\n", upperleft[0], upperleft[1]);
	printf("upperright= %d, %d\n", upperright[0], upperright[1]);
	printf("lowerleft= %d, %d\n", lowerleft[0], lowerleft[1]);
	printf("lowerright= %d, %d\n", lowerright[0], lowerright[1]);
	printf("check_width= %d\n", check_width);
	printf("check_height= %d\n", check_height);

	int center_x=(upperleft[0]+lowerright[0]+upperright[0]+lowerleft[0])/4;
	int center_y=(upperleft[1]+lowerright[1]+upperright[1]+lowerleft[1])/4;

	float cos_theta=getCos(upperleft[0], upperleft[1], upperright[0], upperright[1]);
	float sin_theta=getSin(upperleft[0], upperleft[1], upperright[0], upperright[1]);

	int* device_input;
	int* device_raw_output;
	int* device_resized_output;
	int input_size=3*width*height*sizeof(int);
	int raw_output_size=3*check_height*check_width*sizeof(int);
	int resized_output_size=3*XSIZE*YSIZE*sizeof(int);

	output_resized_image=(int*)malloc(resized_output_size);
	hipMalloc((void**)&device_input, input_size);
	hipMalloc((void**)&device_raw_output, raw_output_size);
	hipMalloc((void**)&device_resized_output, resized_output_size);
	hipMemcpy(device_input, input_image, input_size, hipMemcpyHostToDevice);
	cuCheck(__LINE__);

	dim3 dimBlock1(ceil(check_width/32.0), ceil(check_height/32.0), 1);
	dim3 dimGrid1(32, 32, 1);
	rotation<<<dimGrid1, dimBlock1>>>(device_input, device_raw_output, height, width, check_height, check_width, center_x, center_y, cos_theta, sin_theta);
	cuCheck(__LINE__);

	hipDeviceSynchronize();

	dim3 dimBlock2(ceil(XSIZE/32.0), ceil(YSIZE/32.0), 1);
	dim3 dimGrid2(32, 32, 1);
	resize<<<dimGrid2, dimBlock2>>>(device_raw_output, device_resized_output, check_height, check_width, YSIZE, XSIZE);
	cuCheck(__LINE__);
	hipDeviceSynchronize();

	hipMemcpy(output_resized_image, device_resized_output, resized_output_size, hipMemcpyDeviceToHost);

	hipFree(device_input);
	hipFree(device_raw_output);
	cuCheck(__LINE__);

	ppmWritter("output.pgm", output_resized_image, YSIZE, XSIZE);
	free(input_image);
	free(output_resized_image);
	return device_resized_output;
}


int main() {
	initKNN();

	readSingleCheck(preprocess("testCases/check2.ppm"));

	freeKNN();
	return 0;
}
